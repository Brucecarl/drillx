#include "hip/hip_runtime.h"
#include <stdint.h>
#include <stdio.h>
#include "drillx.h"

extern "C" void hash(uint8_t *challenge, uint8_t *nonce, uint8_t *out) {
    // Allocate device memory for input and output data
    uint8_t *d_challenge, *d_nonce, *d_out;
    hipMalloc((void **)&d_challenge, 32);
    hipMalloc((void **)&d_nonce, 8);
    hipMalloc((void **)&d_out, 16);
	hipMemcpy(d_challenge, challenge, 32, hipMemcpyHostToDevice);
    hipMemcpy(d_nonce, nonce, 8, hipMemcpyHostToDevice);

    // Launch kernel
    do_hash<<<1, 1>>>(d_challenge, d_nonce, d_out);

    // Copy results back to host
    hipMemcpy(out, d_out, 16, hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_challenge);
    hipFree(d_nonce);
    hipFree(d_out);

    // Print errors
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA error: %s\n", hipGetErrorString(err));
    }
}

__global__ void do_hash(uint8_t *d_challenge, uint8_t *d_nonce, uint8_t *d_out) {
    // TODO Run equix code
    *d_out = 42;
}

