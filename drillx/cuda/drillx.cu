#include "hip/hip_runtime.h"
#include <stdint.h>
#include <stdio.h>
#include "drillx.h"
#include "equix.h"
#include "hashx.h"
#include "equix/src/context.h"
#include "equix/src/solver.h"
#include "hashx/src/context.h"

extern "C" void hash(uint8_t *challenge, uint8_t *nonce, uint8_t *out) {
    // Allocate device memory for input and output data
    uint8_t *d_challenge, *d_nonce, *d_out;
    hipMalloc((void **)&d_challenge, 32);
    hipMalloc((void **)&d_nonce, 8);
    hipMalloc((void **)&d_out, 16);
	  hipMemcpy(d_challenge, challenge, 32, hipMemcpyHostToDevice);
    hipMemcpy(d_nonce, nonce, 8, hipMemcpyHostToDevice);

    // Create an equix context
    equix_ctx* ctx = equix_alloc(EQUIX_CTX_SOLVE);
    if (ctx == nullptr) {
        printf("Failed to allocate equix context\n");
        return;
    }

    // Make hashx function
	  if (!hashx_make(ctx->hash_func, challenge, 32)) {
	  	return;
	  }

    // Launch kernel to parallelize hashx operations
    dim3 threadsPerBlock(256); // 256 threads per block
    dim3 blocksPerGrid((65536 + threadsPerBlock.x - 1) / threadsPerBlock.x); // enough blocks to cover 65536 threads
    do_solve_stage0<<<blocksPerGrid, threadsPerBlock>>>(ctx->hash_func, ctx->heap);
    hipDeviceSynchronize();

    // TODO Do the remaining stages
    equix_solution* output = (equix_solution*)malloc(EQUIX_MAX_SOLS * sizeof(equix_solution));
    if (output == NULL) {
        return;
    }
    int sols = solve_stage123(ctx->heap, output);
    printf("sols %d", sols);

    // Free equix context
    equix_free(ctx);

    // Copy results back to host
    hipMemcpy(out, d_out, 16, hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_challenge);
    hipFree(d_nonce);
    hipFree(d_out);

    // Print errors
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA error: %s\n", hipGetErrorString(err));
    }
}

__global__ void do_solve_stage0(hashx_ctx* hash_func, solver_heap* heap) {
    uint16_t i = blockIdx.x * blockDim.x + threadIdx.x;
    prep_stage0(heap);
    if (i < 65536) {
        solve_stage0i(hash_func, heap, i);
    }
}

