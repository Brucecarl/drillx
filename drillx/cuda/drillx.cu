#include "hip/hip_runtime.h"
#include <stdint.h>
#include <stdio.h>
#include "drillx.h"
#include "equix.h"
#include "hashx.h"
#include "equix/src/context.h"
#include "equix/src/solver.h"
#include "equix/src/solver_heap.h"
#include "hashx/src/context.h"
#include <chrono>

// const int BATCH_SIZE = 512;

extern "C" void hash(uint8_t *challenge, uint64_t nonce, uint64_t *out,int batch_size) {
    uint64_t st1=std::chrono::duration_cast<std::chrono::milliseconds>(std::chrono::system_clock::now().time_since_epoch()).count();

    // Generate a hash function for each (challenge, nonce)
    hashx_ctx** ctxs;
    if (hipMallocManaged(&ctxs, batch_size * sizeof(hashx_ctx*)) != hipSuccess) {
        printf("Failed to allocate managed memory for ctxs\n");
        return;
    }
    uint8_t seed[40];
    memcpy(seed, challenge, 32);
    for (int i = 0; i < batch_size; i++) {
        uint64_t nonce_offset = nonce + i;
        memcpy(seed + 32, &nonce_offset, 8);
        ctxs[i] = hashx_alloc(HASHX_INTERPRETED);
        if (!ctxs[i] || !hashx_make(ctxs[i], seed, 40)) {
            //TODO: skip the error batch i if make hashx failed!!
            printf("Failed to make hash:%d,%d\n",nonce_offset,i);
            return;
        }
    }

    uint64_t st=std::chrono::duration_cast<std::chrono::milliseconds>(std::chrono::system_clock::now().time_since_epoch()).count();
    printf("seed time:%d\n",st-st1);
    // Allocate space to hold on to hash values (~500KB per seed)
    uint64_t** hash_space;
    if (hipMallocManaged(&hash_space, batch_size * sizeof(uint64_t*)) != hipSuccess) {
        printf("Failed to allocate managed memory for hash_space\n");
        return;
    }
    for (int i = 0; i < batch_size; i++) {
        if (hipMallocManaged(&hash_space[i], INDEX_SPACE * sizeof(uint64_t)) != hipSuccess) {
            printf("Failed to allocate managed memory for hash_space[%d]\n", i);
            return;
        }
    }

    // Launch kernel to parallelize hashx operations
    dim3 threadsPerBlock(256); // 256 threads per block
    dim3 blocksPerGrid((65536 * batch_size + threadsPerBlock.x - 1) / threadsPerBlock.x); // enough blocks to cover batch
    do_hash_stage0i<<<blocksPerGrid, threadsPerBlock>>>(ctxs, hash_space);
    hipDeviceSynchronize();
    
    uint64_t ht=std::chrono::duration_cast<std::chrono::milliseconds>(std::chrono::system_clock::now().time_since_epoch()).count();
    printf("hash time:%d\n",ht-st);
    // Copy hashes back to cpu
    for (int i = 0; i < batch_size; i++) {
        hipMemcpy(out + i * INDEX_SPACE, hash_space[i], INDEX_SPACE * sizeof(uint64_t), hipMemcpyDeviceToHost);
    }
    uint64_t cpt=std::chrono::duration_cast<std::chrono::milliseconds>(std::chrono::system_clock::now().time_since_epoch()).count();
    printf("copy time:%d\n",cpt-ht);

    // Free memory
    for (int i = 0; i < batch_size; i++) {
        hashx_free(ctxs[i]);
        hipFree(hash_space[i]);
    }
    hipFree(hash_space);
    hipFree(ctxs);

    // Print errors
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA error: %s\n", hipGetErrorString(err));
    }
}

__global__ void do_hash_stage0i(hashx_ctx** ctxs, uint64_t** hash_space) {
    uint32_t item = blockIdx.x * blockDim.x + threadIdx.x;
    uint32_t batch_idx = item / INDEX_SPACE;
    uint32_t i = item % INDEX_SPACE;
    // if (batch_idx < BATCH_SIZE) {
    hash_stage0i(ctxs[batch_idx], hash_space[batch_idx], i);
    // }
}

extern "C" void solve_all_stages(uint64_t *hashes, uint8_t *out, uint32_t *sols) {
    // Create an equix context
    equix_ctx* ctx = equix_alloc(EQUIX_CTX_SOLVE);
    if (ctx == nullptr) {
        printf("Failed to allocate equix context\n");
        return;
    }

    // Do the remaining stages
    equix_solution solutions[EQUIX_MAX_SOLS];
    uint32_t num_sols = equix_solver_solve(hashes, ctx->heap, solutions);

    // Copy results back to host
    memcpy(sols, &num_sols, sizeof(num_sols));
    if (num_sols > 0) {
        memcpy(out, solutions[0].idx, sizeof(solutions[0].idx));
    }

    // Free memory
    equix_free(ctx);
}
